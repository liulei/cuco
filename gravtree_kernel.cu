#include "hip/hip_runtime.h"
#ifndef	_GRAVTREE_KERNEL_H_
#define	_GRAVTREE_KERNEL_H_

#define NEAREST(x) (((x)>boxhalf)?((x)-boxsize):(((x)<-boxhalf)?((x)+boxsize):(x)))

texture<float4, 1, hipReadModeElementType> dPosTex;
texture<float4, 1, hipReadModeElementType> dNodesTex;
texture<int, 1, hipReadModeElementType> dNextnodeTex;

__constant__ SIMPARAM	dSimParam;

__constant__ SOFTPARAM	dSoftParam;

__device__	float4	*Pos;

//__global__ void force_treebuild_device(
//					float4)

__global__ void force_treeevaluate_shortrange_device(
						float4	*dPos,
						float4	*dGravAccel,
						NODE	*dNodes,
						EXTNODE	*dExtnodes,
						int		*dNextnode,
						int		*dFather,
						int 	numParticles){

	uint	index	=	blockIdx.y * gridDim.x * blockDim.x
						+ blockIdx.x * blockDim.x + threadIdx.x;
	if(index >= numParticles){
		return;
	}
	
	Pos	=	dPos;

//	NODE	node;
	NODE_1	node_1;
	NODE_2	node_2;
	NODE_3	node_3;
	int		no, tabindex;
	float	r2, dx, dy, dz, r, fac, u;
	float	eff_dist, dist;
	float4	pos, pos_no, acc;

	float4	temp;
	
	float	rcut	=	dSimParam.rcut;
	float	rcut2	=	dSimParam.rcut2;
	float	asmthfac	=	dSimParam.asmthfac;
	float	mass	=	dSimParam.mass;
	float	ErrTolTheta	=	dSimParam.ErrTolTheta;
	float	G		=	dSimParam.G;
	float	boxsize	=	dSimParam.boxsize;
	float	boxhalf	=	boxsize / 2.0;

/*
	printf("device parameter: \n");
	printf("rcut: %g\n", rcut);
	printf("rcut2: %g\n", rcut2);
	printf("asmthfac: %g\n", asmthfac);
	printf("mass: %g\n", mass);
	printf("ErrTolTheta: %g\n", ErrTolTheta);
	printf("G: %g\n", G);
	printf("boxsize: %g\n", boxsize);
	printf("boxhalf: %g\n", boxhalf);
*/
	float	h	=	dSoftParam.h;
	float	h_inv	=	dSoftParam.h_inv;
	float	h3_inv	=	dSoftParam.h3_inv;
/*
	printf("h: %g\n", h);
	printf("h_inv: %g\n", h_inv);
	printf("h3_inv: %g\n", h3_inv);
*/
	acc.x	=	0.0;
	acc.y	=	0.0;
	acc.z	=	0.0;

	pos	=	Pos[index];
	
	no	=	numParticles;

	while(no >= 0){

		if(no < numParticles){

//			pos_no	=	dPos[no];
			pos_no	=	tex1Dfetch(dPosTex, no);
			
			dx	=	pos_no.x - pos.x;
			dy	=	pos_no.y - pos.y;
			dz	=	pos_no.z - pos.z;

			dx	=	NEAREST(dx);
			dy	=	NEAREST(dy);
			dz	=	NEAREST(dz);

			r2	=	dx * dx + dy * dy + dz * dz;

			mass	=	dSimParam.mass;

//			no	=	dNextnode[no];
			no	=	tex1Dfetch(dNextnodeTex, no);
		
		}else{

//			node	=	dNodes[no];
//			node	=	tex1Dfetch(dNodesTex, no);

			temp	=	tex1Dfetch(dNodesTex, 3 * (no - numParticles));
			node_1	=	*(NODE_1 *) &temp;

			temp	=	tex1Dfetch(dNodesTex, 3 * (no - numParticles) + 1);
			node_2	=	*(NODE_2 *) &temp;

			temp	=	tex1Dfetch(dNodesTex, 3 * (no - numParticles) + 2);
			node_3	=	*(NODE_3 *) &temp;

			mass	=	node_2.mass;
			
			dx	=	node_2.s[0] - pos.x;
			dy	=	node_2.s[1] - pos.y;
			dz	=	node_2.s[2] - pos.z;

			dx	=	NEAREST(dx);
			dy	=	NEAREST(dy);
			dz	=	NEAREST(dz);

			r2	=	dx * dx + dy * dy + dz * dz;

			if(r2 > rcut2){

				eff_dist	=	rcut + 0.5 * node_1.len;

				dist	=	NEAREST(node_1.center[0] - pos.x);
				if(dist < -eff_dist || dist > eff_dist){
					no	=	node_3.sibling;
					continue;
				}

				dist	=	NEAREST(node_1.center[1] - pos.y);
				if(dist < -eff_dist || dist > eff_dist){
					no	=	node_3.sibling;
					continue;
				}

				dist	=	NEAREST(node_1.center[2] - pos.z);
				if(dist < -eff_dist || dist > eff_dist){
					no	=	node_3.sibling;
					continue;
				}
			}
			
			if(node_1.len * node_1.len > r2 * ErrTolTheta * ErrTolTheta){
				no	=	node_3.nextnode;
				continue;
			}

			no	=	node_3.sibling;
		}

		r	=	__fsqrt_rn(r2);

		if(r >= h){
			fac	=	mass / (r2 * r);
		}else{
			u	=	r * h_inv;
			if(u < 0.5)
				fac	=	mass * h3_inv * (10.66667 + u * u * (32.0 * u - 38.4));
			else
				fac	=	mass * h3_inv * (21.33333 - 48.0 * u 
				+ 38.4 * u * u - 10.66667 * u * u * u - 0.06667 / (u * u * u));
		}

		tabindex	=	(int) (asmthfac * r);

		if(tabindex < NTAB){

			fac	*=	dSimParam.shortrange_table[tabindex];

			acc.x	+=	dx * fac;
			acc.y	+=	dy * fac;
			acc.z	+=	dz * fac;
		}
	}
	
	acc.x	*=	G;
	acc.y	*=	G;
	acc.z	*=	G;

	dGravAccel[index]	=	acc;

}

#endif
