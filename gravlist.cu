#include "hip/hip_runtime.h"

#include	<stdio.h>
#include	<stdlib.h>
#include	<string.h>
#include	<math.h>

#include	<cutil_inline.h>

#include	<cutil_math.h>

#include	"allvars.h"
#include	"proto.h"

#include	"radixsort.h"
#include	"proto.cuh"
#include	"gravlist_kernel.cu"

extern "C"{

RadixSort	*sorter;

void cudaInit(){
	
	printf("Init cuda device...\n");

	hipSetDevice( cutGetMaxGflopsDeviceId());

}

void allocateArray(void **devPtr, size_t size){

	cutilSafeCall(hipMalloc(devPtr, size));

}

void freeArray(void *devPtr){

	cutilSafeCall(hipFree(devPtr));

}

void copyArrayFromDevice(void *host, void *device, int size){

	cutilSafeCall(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));

}

void copyArrayToDevice(void *device, void *host, int offset, int size){
	cutilSafeCall(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));

}

void copyPosToDevice(){
	
	float4	*hPos4	=	(float4 *)hPos;

	uint	i;
	for(i = 0; i < NumPart; ++i){

		hPos4[i].x	=	P[i].Pos[0];
		hPos4[i].y	=	P[i].Pos[1];
		hPos4[i].z	=	P[i].Pos[2];
	}
	copyArrayToDevice(dOldPos, hPos, 0, numParticles * sizeof(float4));
}

void copyAccelFromDevice(){
	
	copyArrayFromDevice(hGravAccel, dGravAccel, numParticles * sizeof(float4));

	float4	*hGravAccel4	=	(float4 *)hGravAccel;

	int	i;
	for(i = 0; i < NumPart; ++i){

		P[i].GravAccel[0]	=	hGravAccel4[i].x;
		P[i].GravAccel[1]	=	hGravAccel4[i].y;
		P[i].GravAccel[2]	=	hGravAccel4[i].z;
	}
}

void calcHash(uint * gridParticleHash,
			  uint * gridParticleIndex,
			  float *pos,
			  uint	numParticles){

	uint	numThreads, numBlocks;

	numThreads	=	256;
	numBlocks	=	numParticles / numThreads;

	calcHashD<<<numBlocks, numThreads>>>(gridParticleHash,
										 gridParticleIndex,
										 (float4 *) pos,
										 numParticles);
}



void reorderDataAndFindCellStart(uint 	*cellStart,
								 uint 	*cellEnd,
								 float	*sortedPos,
								 uint	*gridParticleHash,
								 uint	*gridParticleIndex,
								 float	*oldPos,
								 uint	numParticles,
								 uint	numCells){
	
	uint	numThreads, numBlocks;
	
	numThreads	=	256;
	numBlocks	=	numParticles / numThreads;

	cutilSafeCall(hipMemset(cellStart, 0xffffffff, numCells * sizeof(uint)));

	cutilSafeCall(hipBindTexture(0, oldPosTex, oldPos, numParticles * sizeof(float4)));

	uint	smemSize	=	sizeof(uint) * (numThreads + 1);

	reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(
		cellStart,
		cellEnd,
		(float4 *) sortedPos,
		gridParticleHash,
		gridParticleIndex,
		(float4 *) oldPos,
		numParticles);

	cutilSafeCall(hipUnbindTexture(oldPosTex));
}

void cudaForceEvaluateShortrange(float	*gravAccel,
								 float	*sortedPos,
								 uint	*gridParticleIndex,
								 uint	*cellStart,
								 uint	*cellEnd,
								 uint	numParticles,
								 uint	numCells){
	cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPos, numParticles * sizeof(float4)));
	cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numCells * sizeof(uint)));
	cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numCells * sizeof(uint)));

	uint	numThreads, numBlocks;

	numThreads	=	256;
	numBlocks	=	numParticles / numThreads;

	cudaForceEvaluateShortrangeD<<<numBlocks, numThreads>>>((float4 *)gravAccel,
															(float4 *)sortedPos,
															gridParticleIndex,
															cellStart,
															cellEnd,
															numParticles);
}

void setSimParam(SIMPARAM *pHostSimParam){

	cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dSimParam), pHostSimParam, sizeof(SIMPARAM)));

}

void setSoftParam(SOFTPARAM *pHostSoftParam){
	cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dSoftParam), pHostSoftParam, sizeof(SOFTPARAM)));
}

}
